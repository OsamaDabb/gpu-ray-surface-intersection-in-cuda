#include "hip/hip_runtime.h"
// WE'RE FIXING THIS VERSION: 
// - get rid of expensive i/o at each iteration
// - avoid recreating bvh at each iteration
// - make a class with a C interface for python access
// readFile, writeFile are the problem
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <iterator>
#include <typeinfo>
#include <vector>

#include <stdint.h>
#include "bvh_structure.h"
#include "rsi_geometry.h"

using namespace std;
using namespace lib_bvh;
using namespace lib_rsi;

#ifdef __cplusplus
extern "C" {
#endif

// Pure C-style declarations here
void* setup_RSI(float* vertices, int* triangles, int num_vertices, int num_triangles, int num_rays);
void detect_RSI(void* rsi_obj, float* rayFrom, float* rayTo, int** out_intersectTriangle, float** out_baryT);
void destroy_RSI(void* rsi_obj);
static void HandleError(hipError_t err, const char *file, int line);


#ifdef __cplusplus
}
#endif

//-------------------------------------------------
// This implementation corresponds to version v3
// with support for barycentric mode and the
// intercept_count experimental feature
//-------------------------------------------------

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void CheckSyncAsyncErrors(const char* file, int line)
{
    // Inspired from https://developer.nvidia.com/blog/how-query-device-properties-and-handle-errors-cuda-cc/
    hipError_t errSync = hipGetLastError(); // returns the value of the latest asynchronous error and also resets it to hipSuccess.
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
    {
        printf("Sync kernel error\n");
        HandleError(errSync, file, line);
    }
    if (errAsync != hipSuccess)
    {
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
        HandleError(errAsync, file, line);
    }
}

#define CUDA_SYNCHRO_CHECK() (CheckSyncAsyncErrors(__FILE__, __LINE__))

template <class T>
int readData(string fname, vector<T> &v, int dim=1, bool silent=false)
{
    ifstream infile(fname.c_str(), ios::binary | ios::ate);
    if (! infile) {
        cerr << "File " << fname << " not found" << endl;
        exit(1);
    }
    ifstream::pos_type nbytes = infile.tellg();
    infile.seekg(0, infile.beg);
    const int elements = nbytes / sizeof(T);
    v.resize(elements);
    infile.read(reinterpret_cast<char*>(v.data()), nbytes);
    if (! silent) {
        cout << fname << " contains " << nbytes << " bytes, "
             << v.size() << " <" << typeid(v.front()).name() << ">, "
             << v.size() / dim << " elements" << endl;
    }
    return elements / dim;
}

template <class T>
void writeData(string fname, vector<T> &v)
{
    ofstream outfile(fname.c_str(), ios::out | ios::binary);
    if (! outfile) {
        cerr << "Cannot create " << fname << " for writing" << endl;
        exit(1);
    }
    outfile.write(reinterpret_cast<char*>(v.data()), v.size() * sizeof(T));
    outfile.close();
}


// OsamaDabb functions etc.
inline void safeCudaFree(void* ptr)
{
    if (ptr != nullptr)
    {
        HANDLE_ERROR(hipFree(ptr));
        ptr = nullptr;  // Optional: prevents double-free later
    }
}

// End OsamaDabb


class RSI {
    public:
        const bool checkEnabled;
        const float largePosVal;
        const bool barycentric;
        const bool quietMode;

        vector<float> h_vertices;
        vector<int>   h_triangles;
        vector<float> h_rayFrom;
        vector<float> h_rayTo;
        vector<int>   h_crossingDetected;
        vector<int>   h_intersectTriangle;
        vector<float> h_baryT, h_baryU, h_baryV;
        vector<uint64_t> h_morton;
        vector<int> h_sortedTriangleIDs;
        int nVertices, nTriangles, nRays;

        /// Device pointers
        float* d_vertices = nullptr;
        int* d_triangles = nullptr;
        float* d_rayFrom = nullptr;
        float* d_rayTo = nullptr;
        AABB* d_rayBox = nullptr;
        int* d_crossingDetected = nullptr;
        int* d_intersectTriangle = nullptr;
        float* d_baryT = nullptr;
        float* d_baryU = nullptr;
        float* d_baryV = nullptr;

        BVHNode* d_leafNodes = nullptr;
        BVHNode* d_internalNodes = nullptr;
        uint64_t* d_morton = nullptr;
        int* d_sortedTriangleIDs = nullptr;
        CollisionList* d_hitIDs = nullptr;
        InterceptDistances* d_interceptDists = nullptr;

        // Grid sizes
        int blockX;
        int gridXr;
        int gridXt;
        int gridXLambda;

        // CUDA timing
        hipEvent_t start, end;

        // Sizes
        int sz_vertices;
        int sz_triangles;
        int sz_rays;
        int sz_rbox;
        int sz_id;
        int sz_bary;
        int sz_morton;
        int sz_sortedIDs;
        int sz_hitIDs;
        int sz_interceptDists;

        // Extents
        float minval[3];
        float maxval[3];
        float half_delta[3];
        float inv_delta[3];


        // Constructor to initialize consts
        RSI()
            : checkEnabled(true), largePosVal(2.5e8f), barycentric(true),
            quietMode(true),
            d_vertices(nullptr),d_triangles(nullptr),
            d_rayFrom(nullptr),d_rayTo(nullptr),
            d_rayBox(nullptr),
            d_crossingDetected(nullptr),
            d_intersectTriangle(nullptr),
            d_baryT(nullptr),
            d_baryU(nullptr),
            d_baryV(nullptr),
            d_leafNodes(nullptr),
            d_internalNodes(nullptr),
            d_morton(nullptr),
            d_sortedTriangleIDs(nullptr),
            d_hitIDs(nullptr),
            d_interceptDists(nullptr),
            blockX(0),gridXr(0),gridXt(0),
            gridXLambda(0),
            sz_vertices(0),
            sz_triangles(0),
            sz_rays(0),
            sz_rbox(0),
            sz_id(0),
            sz_bary(0),
            sz_morton(0),
            sz_sortedIDs(0),
            sz_hitIDs(0),
            sz_interceptDists(0)
        {
            // Initialize minval, maxval, half_delta, inv_delta arrays
            for (int i = 0; i < 3; ++i) {
                minval[i] = 0.0f;
                maxval[i] = 0.0f;
                half_delta[i] = 0.0f;
                inv_delta[i] = 0.0f;
            }
        }

        void setup(float* in_vertices, int* in_triangles, int num_vertices, int num_triangles, int num_rays){
            
            h_vertices.assign(in_vertices, in_vertices + 3 * num_vertices);
            h_triangles.assign(in_triangles, in_triangles + 3*num_triangles);
            nVertices = num_vertices;
            nTriangles = num_triangles;
            nRays = num_rays;

            sz_vertices = 3 * nVertices * sizeof(float);
            sz_triangles = 3 * nTriangles * sizeof(int);
            sz_rays = 3 * nRays * sizeof(float);
            sz_rbox = nRays * sizeof(AABB);
            sz_id = nRays * sizeof(int);
            sz_bary = nRays * sizeof(float);

            h_crossingDetected.resize(nRays);
         
            HANDLE_ERROR(hipMalloc(&d_vertices, sz_vertices));
            HANDLE_ERROR(hipMalloc(&d_triangles, sz_triangles));
            HANDLE_ERROR(hipMalloc(&d_rayFrom, sz_rays));
            HANDLE_ERROR(hipMalloc(&d_rayTo, sz_rays));
            HANDLE_ERROR(hipMalloc(&d_rayBox, sz_rbox));
        
            h_intersectTriangle.resize(nRays);
            h_baryT.resize(nRays);
            h_baryU.resize(nRays);
            h_baryV.resize(nRays);
            HANDLE_ERROR(hipMalloc(&d_intersectTriangle, sz_id));
            HANDLE_ERROR(hipMalloc(&d_baryT, sz_bary));
            HANDLE_ERROR(hipMalloc(&d_baryU, sz_bary));
            HANDLE_ERROR(hipMalloc(&d_baryV, sz_bary));
                
            HANDLE_ERROR(hipMemcpy(d_vertices, h_vertices.data(), sz_vertices, hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(d_triangles, h_triangles.data(), sz_triangles, hipMemcpyHostToDevice));

            //grid partitions
            blockX = 1024,
            gridXr = (int)ceil((float)nRays / blockX),
            gridXt = (int)ceil((float)nTriangles / blockX),
            gridXLambda = 16; //N_{grids}
            if (! quietMode) {
                cout << blockX << " threads/block, grids: {triangles: "
                    << gridXt << ", rays: " << gridXLambda << "}" << endl;
            }

            //order triangles using Morton code
            //- normalise surface vertices to canvas coords
            getMinMaxExtentOfSurface<float>(h_vertices, minval, maxval, half_delta,
                inv_delta, nVertices, quietMode);
            //- convert centroid of triangles to morton code
            createMortonCode<float, uint64_t>(h_vertices, h_triangles,
                        minval, half_delta, inv_delta,
                        h_morton, nTriangles);
            //- sort before constructing binary radix tree
            sortMortonCode<uint64_t>(h_morton, h_sortedTriangleIDs);
            if (!quietMode && checkEnabled) {
                cout << "checking sortMortonCode" << endl;
                for (int j = 0; j < min(12, nTriangles); j++) {
                    cout << j << ": (" << h_sortedTriangleIDs[j] << ") "
                    << h_morton[j] << endl;
                }
            }


            sz_morton = nTriangles * sizeof(uint64_t);
            sz_sortedIDs = nTriangles * sizeof(int);
            sz_hitIDs = gridXLambda * blockX * sizeof(CollisionList);
            sz_interceptDists = gridXLambda * blockX * sizeof(InterceptDistances);
            //data structures used in agglomerative LBVH construction
            HANDLE_ERROR(hipMalloc(&d_leafNodes, nTriangles * sizeof(BVHNode)));
            HANDLE_ERROR(hipMalloc(&d_internalNodes, nTriangles * sizeof(BVHNode)));
            HANDLE_ERROR(hipMalloc(&d_morton, sz_morton));
            HANDLE_ERROR(hipMalloc(&d_sortedTriangleIDs, sz_sortedIDs));
            HANDLE_ERROR(hipMalloc(&d_hitIDs, sz_hitIDs));

            HANDLE_ERROR(hipMemcpy(d_morton, h_morton.data(), sz_morton, hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(d_sortedTriangleIDs, h_sortedTriangleIDs.data(), sz_sortedIDs, hipMemcpyHostToDevice));
            std::vector<uint64_t>().swap(h_morton);
            std::vector<int>().swap(h_sortedTriangleIDs);

            bvhResetKernel<<<gridXt, blockX>>>(d_vertices, d_triangles,
                d_internalNodes, d_leafNodes,
                d_sortedTriangleIDs, nTriangles);
            HANDLE_ERROR(hipDeviceSynchronize());

            bvhConstruct<uint64_t><<<gridXt, blockX>>>(d_internalNodes, d_leafNodes,
                                    d_morton, nTriangles);
            HANDLE_ERROR(hipDeviceSynchronize());
            CUDA_SYNCHRO_CHECK();

        }

    void detect(float* rayFrom, float* rayTo){

        h_rayFrom.assign(rayFrom, rayFrom + 3*nRays);
        h_rayTo.assign(rayTo, rayTo + 3 * nRays);
        
        HANDLE_ERROR(hipMemcpy(d_rayFrom, h_rayFrom.data(), sz_rays, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(d_rayTo, h_rayTo.data(), sz_rays, hipMemcpyHostToDevice));

        // BVH BUILDING CODE, maybe relies on ray data to create

        //initialise arrays
        initArrayKernel<<<gridXr, blockX>>>(d_intersectTriangle, -1, nRays);
        initArrayKernel<<<gridXr, blockX>>>(d_baryT, largePosVal, nRays);

        HANDLE_ERROR(hipDeviceSynchronize());

        //compute ray-segment bounding boxes
        rbxKernel<<<gridXr, blockX>>>(d_rayFrom, d_rayTo, d_rayBox, nRays);
        HANDLE_ERROR(hipDeviceSynchronize());

        // END BVH CODE

        bvhIntersectionKernel<<<gridXLambda, blockX>>>(
                    d_vertices, d_triangles, d_rayFrom, d_rayTo,
                    d_internalNodes, d_rayBox, d_hitIDs,
                    d_intersectTriangle, d_baryT, d_baryU, d_baryV,
                    nTriangles, nRays);
                    
        HANDLE_ERROR(hipDeviceSynchronize());
    
        HANDLE_ERROR(hipMemcpy(h_intersectTriangle.data(), d_intersectTriangle,
                                sz_id, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryT.data(), d_baryT, sz_bary, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryU.data(), d_baryU, sz_bary, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryV.data(), d_baryV, sz_bary, hipMemcpyDeviceToHost));

        // printf("Detected triangle ID = %d\n", h_intersectTriangle[0]);
    
    }

    void destroy(){
        
        safeCudaFree(d_vertices);
        safeCudaFree(d_triangles);
        safeCudaFree(d_rayFrom);
        safeCudaFree(d_rayTo);
        safeCudaFree(d_rayBox);
        safeCudaFree(d_intersectTriangle);
        safeCudaFree(d_baryT);
        safeCudaFree(d_baryU);
        safeCudaFree(d_baryV);
        safeCudaFree(d_leafNodes);
        safeCudaFree(d_internalNodes);
        safeCudaFree(d_morton);
        safeCudaFree(d_sortedTriangleIDs);
        safeCudaFree(d_hitIDs);

    }
};

// int main(){
//     RSI* rsi = new RSI();

//     // 9 vertices (3 triangles, 3 vertices each)
//     float vertices_array[27] = {
//         // Triangle 0
//         0.0f, 0.0f, 0.0f,   // Vertex 0
//         1.0f, 0.0f, 0.0f,   // Vertex 1
//         0.0f, 1.0f, 0.0f,   // Vertex 2

//         // Triangle 1
//         2.0f, 0.0f, 0.0f,   // Vertex 3
//         3.0f, 0.0f, 0.0f,   // Vertex 4
//         2.0f, 1.0f, 0.0f,   // Vertex 5

//         // Triangle 2
//         4.0f, 0.0f, 0.0f,   // Vertex 6
//         5.0f, 0.0f, 0.0f,   // Vertex 7
//         4.0f, 1.0f, 0.0f    // Vertex 8
//     };

//     int triangles_array[9] = {
//         0, 1, 2,  // Triangle 0
//         3, 4, 5,  // Triangle 1
//         6, 7, 8   // Triangle 2
//     };

//     // Pointers
//     float* vertices = vertices_array;
//     int* triangles = triangles_array;

//     // Number of primitives
//     int num_vertices = 9;    // 9 vertices
//     int num_triangles = 3;   // 3 triangles
//     int num_rays = 1;        // 1 ray

//     // Define a ray that will hit the first triangle
//     float rayFrom_array[3] = { 0.5f, 0.5f, 1.0f };   // Above Triangle 0
//     float rayTo_array[3]   = { 0.5f, 0.5f, -1.0f };  // Going down through Triangle 0

//     // Setup
//     rsi->setup(vertices, triangles, num_vertices, num_triangles, num_rays);

//     // Detect
//     rsi->detect(rayFrom_array, rayTo_array);

//     // Optionally you might want to inspect rsi->h_intersectTriangle, rsi->h_baryT here
//     // Example:
//     printf("Intersected Triangle ID: %d\n", rsi->h_intersectTriangle[0]);
//     printf("Barycentric T: %f\n", rsi->h_baryT[0]);

//     // Destroy
//     rsi->destroy();

//     delete rsi;
//     return 0;
// }


// Create RSI object and call setup
extern "C" void* setup_RSI(float* vertices, int* triangles, int num_vertices, int num_triangles, int num_rays) {
    RSI* rsi = new RSI();
    rsi->setup(vertices, triangles, num_vertices, num_triangles, num_rays);
    return static_cast<void*>(rsi);
}

// Call detect on an existing RSI object
extern "C" void detect_RSI(void* rsi_obj, float* rayFrom, float* rayTo, int** out_intersectTriangle, float** out_baryT) {
    RSI* rsi = static_cast<RSI*>(rsi_obj);
    rsi->detect(rayFrom, rayTo);

    // printf("rayFrom = (%f, %f, %f)\n", rayFrom[0], rayFrom[1], rayFrom[2]);
    // printf("rayTo = (%f, %f, %f)\n", rayTo[0], rayTo[1], rayTo[2]);
    
    *out_intersectTriangle = rsi->h_intersectTriangle.data();
    *out_baryT = rsi->h_baryT.data();
}

// Destroy RSI object
extern "C" void destroy_RSI(void* rsi_obj) {
    RSI* rsi = static_cast<RSI*>(rsi_obj);
    delete rsi;
}


    